#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include "proto.h"
#define HISTSIZE 256

void freeCuda(int* arr)
{
	hipError_t err = hipFree(arr);
	if (err != hipSuccess) {
    		fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
            	hipGetErrorString(err));
    		exit(EXIT_FAILURE);
	}
}

__global__ void histArr(const int* mainArr,int* d_temp, int numOfElementsPerThread, int range)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
  	int offset_Arr = id*numOfElementsPerThread;  // Start of the part of mainArr for this thread
  	int offset_temp = id*range; // Start of the part of temp for this thread
  	int index;

  	// Jump to the place in main array and update the proper part of the temp array
  	for (int i = 0;   i < numOfElementsPerThread;  i++) {
    		 index = mainArr[offset_Arr + i];
   		 d_temp[offset_temp + index]++;
  	}	
}


// Initialize the temporary array
__global__ void initTemp(int* arr) {

  int i;
  int thread_id = blockIdx.x*blockDim.x+threadIdx.x;  
  for (i = 0;   i < HISTSIZE;   i++)   // Each of 200 threads initialize 256 members in temp
    arr[thread_id*HISTSIZE + i] = 0;  

}

// Unify all values in the temp
__global__ void collectResults(int *d_temp, int *d_histo, int part) {
  
  int index = threadIdx.x;  
  int result = 0, i;
  for (i = 0;  i < part; i++) {
  	result += d_temp[index + HISTSIZE*i];  	
  }
  d_histo[index] = result;

}

int startCudaTask(int *h_mainArr, int* h_hist, int numElements, int range) {
	
	hipError_t err = hipSuccess;
	int threadsPerBlock = 20;
	int blocksPerGrid = 10;
	
        size_t size = numElements * sizeof(int);   
        
  	int *d_mainArr = NULL;
  	err = hipMalloc((void **)&d_mainArr, size);
  	if (err != hipSuccess) {
   		 fprintf(stderr, "Failed to allocate device d_mainArr (error code %s)!\n",
           		 hipGetErrorString(err));
    		 exit(EXIT_FAILURE);
  	}
  	
	//Temp for histograms for each thread - each thread will calculate it part 
        int *d_temp = NULL;
  	err = hipMalloc((void **)&d_temp, HISTSIZE*threadsPerBlock*blocksPerGrid*sizeof(int));
  	if (err != hipSuccess) {
    		fprintf(stderr, "Failed to allocate device d_temp(error code %s)!\n",
        	    hipGetErrorString(err));
   		exit(EXIT_FAILURE);
  	}

  	// Allocate histo on device
  	int *d_hist = NULL;
  	err = hipMalloc((void **)&d_hist, HISTSIZE*sizeof(int));
  	if (err != hipSuccess) {
    		fprintf(stderr, "Failed to allocate device d_hist (error code %s)!\n",
            		hipGetErrorString(err));
    		exit(EXIT_FAILURE);
  	}

  	err = hipMemcpy(d_mainArr, h_mainArr, size, hipMemcpyHostToDevice);
  	  	
  	initTemp<<<blocksPerGrid, threadsPerBlock>>>(d_temp);
  	err = hipGetLastError();
      	if (err != hipSuccess) {
    		fprintf(stderr, "Failed to launch initTemp kernel (error code %s)!\n",
            		hipGetErrorString(err));
    		exit(EXIT_FAILURE);
  	}

     	//Calculate the part for each thread in main array
  	int elementsPerThread = numElements/(threadsPerBlock*blocksPerGrid);
  	histArr<<<blocksPerGrid, threadsPerBlock>>>(d_mainArr, d_temp, elementsPerThread, HISTSIZE);
	err = hipGetLastError();
	if (err != hipSuccess) {
    		fprintf(stderr, "Failed to launch histArr kernel (error code %s)!\n",
          	  hipGetErrorString(err));
    		exit(EXIT_FAILURE);
	}
	//hipDeviceSynchronize();
	collectResults<<<1, HISTSIZE>>>(d_temp, d_hist, threadsPerBlock*blocksPerGrid);
	err = hipGetLastError();
	if (err != hipSuccess) {
    		fprintf(stderr, "Failed to launch collectResults kernel (error code %s)!\n",
          	  hipGetErrorString(err));
    		exit(EXIT_FAILURE);
	}

	// Copy the  result from GPU to the host memory.
        err = hipMemcpy(h_hist, d_hist, HISTSIZE*sizeof(int), hipMemcpyDeviceToHost);
  	if (err != hipSuccess) {
    		fprintf(stderr, "133: Failed to copy memory from device to host (error code %s)!\n",
            		hipGetErrorString(err));     			
            	exit(EXIT_FAILURE);
  	}  
  		  
  	freeCuda(d_mainArr);
  	freeCuda(d_temp);
  	freeCuda(d_hist);
  	return 1;
}


